
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {

__global__
void set_consecutive_sequence(
    uint32_t num_d_out,
    uint32_t* d_out)
{
    int i_d_out = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_d_out >= num_d_out ){ return; }
    //
    d_out[i_d_out] = i_d_out;
}

__global__
void shift_array_right(
    uint32_t n,
    uint32_t* din,
    uint32_t* dout
)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= n ){ return; }
    //
    if( i == 0 ){ return; }
    dout[i] = din[i-1];
}

__global__
void permute(
    unsigned int n,
    uint32_t* new2data,
    const uint32_t* new2old,
    const uint32_t* old2data)
{
    int i_new = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_new >= n ){ return; }
    //
    int i_old = new2old[i_new];
    assert(i_new < n);
    new2data[i_new] = old2data[i_old];
}

__global__
void set_value_at_mask(
    unsigned int n,
    float* elem2value,
    float set_value,
    const uint32_t* elem2mask,
    uint32_t mask,
    bool is_set_value_at_mask_value_equal)
{
    int i_elem = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_elem >= n ){ return; }
    //
    if( (elem2mask[i_elem] == mask) == is_set_value_at_mask_value_equal) {
        elem2value[i_elem] = set_value;
    }
}

__global__
void sort_indexed_array(
    unsigned int n,
    const uint32_t* p2idx,
    uint32_t* idx2q
)
{
    int p = blockDim.x * blockIdx.x + threadIdx.x;
    if( p >= n ){ return; }
    //
    const uint32_t idx0 = p2idx[p];
    const uint32_t idx1 = p2idx[p+1];
    for(uint32_t idx=idx0;idx<idx1;++idx){
        uint32_t idx_min = idx;
        for(uint32_t jdx=idx+1;jdx<idx1;++jdx){
            if( idx2q[jdx] < idx2q[idx_min] ){
                idx_min = jdx;
            }
        }
        const uint32_t tmp = idx2q[idx];
        idx2q[idx] = idx2q[idx_min];
        idx2q[idx_min] = tmp;
    }

}


}