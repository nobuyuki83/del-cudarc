
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {

__global__
void set_consecutive_sequence(
    uint32_t num_d_out,
    uint32_t* d_out)
{
    int i_d_out = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_d_out >= num_d_out ){ return; }
    //
    d_out[i_d_out] = i_d_out;
}

__global__
void shift_array_right(
    uint32_t n,
    uint32_t* din,
    uint32_t* dout
)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= n ){ return; }
    //
    if( i == 0 ){ return; }
    dout[i] = din[i-1];
}

__global__
void permute(
    unsigned int num_new,
    uint32_t* new2data,
    const uint32_t* new2old,
    const uint32_t* old2data)
{
    int i_new = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_new >= num_new ){ return; }
    //
    int i_old = new2old[i_new];
    new2data[i_new] = old2data[i_old];
}

__global__
void set_value_at_mask(
    unsigned int n,
    float* elem2value,
    float set_value,
    const uint32_t* elem2mask,
    uint32_t mask,
    bool is_set_value_at_mask_value_equal)
{
    int i_elem = blockDim.x * blockIdx.x + threadIdx.x;
    if( i_elem >= n ){ return; }
    //
    if( (elem2mask[i_elem] == mask) == is_set_value_at_mask_value_equal) {
        elem2value[i_elem] = set_value;
    }
}

}