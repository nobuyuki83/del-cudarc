
#include <hip/hip_runtime.h>
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
	((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

extern "C" {

// Modified version of Mark Harris' implementation of the Blelloch scan
//  according to https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
__global__
void gpu_prescan(unsigned int* const d_out,
	const unsigned int* const d_in,
	unsigned int* const d_block_sums,
	const unsigned int len,
	const unsigned int shmem_sz,
	const unsigned int max_elems_per_block)
{
	// Allocated on invocation
	extern __shared__ unsigned int s_out[];

	int thid = threadIdx.x;
	int ai = thid;
	int bi = thid + blockDim.x;

	// Zero out the shared memory
	// Helpful especially when input size is not power of two
	s_out[thid] = 0;
	s_out[thid + blockDim.x] = 0;
	// If CONFLICT_FREE_OFFSET is used, shared memory
	//  must be a few more than 2 * blockDim.x
	if (thid + max_elems_per_block < shmem_sz)
		s_out[thid + max_elems_per_block] = 0;

	__syncthreads();

	// Copy d_in to shared memory
	// Note that d_in's elements are scattered into shared memory
	//  in light of avoiding bank conflicts
	unsigned int cpy_idx = max_elems_per_block * blockIdx.x + threadIdx.x;
	if (cpy_idx < len)
	{
		s_out[ai + CONFLICT_FREE_OFFSET(ai)] = d_in[cpy_idx];
		if (cpy_idx + blockDim.x < len)
			s_out[bi + CONFLICT_FREE_OFFSET(bi)] = d_in[cpy_idx + blockDim.x];
	}

	// For both upsweep and downsweep:
	// Sequential indices with conflict free padding
	//  Amount of padding = target index / num banks
	//  This "shifts" the target indices by one every multiple
	//   of the num banks
	// offset controls the stride and starting index of
	//  target elems at every iteration
	// d just controls which threads are active
	// Sweeps are pivoted on the last element of shared memory

	// Upsweep/Reduce step
	int offset = 1;
	for (int d = max_elems_per_block >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * ((thid << 1) + 1) - 1;
			int bi = offset * ((thid << 1) + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			s_out[bi] += s_out[ai];
		}
		offset <<= 1;
	}

	// Save the total sum on the global block sums array
	// Then clear the last element on the shared memory
	if (thid == 0)
	{
		d_block_sums[blockIdx.x] = s_out[max_elems_per_block - 1
			+ CONFLICT_FREE_OFFSET(max_elems_per_block - 1)];
		s_out[max_elems_per_block - 1
			+ CONFLICT_FREE_OFFSET(max_elems_per_block - 1)] = 0;
	}

	// Downsweep step
	for (int d = 1; d < max_elems_per_block; d <<= 1)
	{
		offset >>= 1;
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * ((thid << 1) + 1) - 1;
			int bi = offset * ((thid << 1) + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			unsigned int temp = s_out[ai];
			s_out[ai] = s_out[bi];
			s_out[bi] += temp;
		}
	}
	__syncthreads();

	// Copy contents of shared memory to global memory
	if (cpy_idx < len)
	{
		d_out[cpy_idx] = s_out[ai + CONFLICT_FREE_OFFSET(ai)];
		if (cpy_idx + blockDim.x < len)
			d_out[cpy_idx + blockDim.x] = s_out[bi + CONFLICT_FREE_OFFSET(bi)];
	}
}


__global__
void gpu_add_block_sums(uint32_t* const d_out,
	const uint32_t* const d_block_sums,
	const uint32_t numElems)
{
	//unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int d_block_sum_val = d_block_sums[blockIdx.x];
	//printf("%d %d %d\n", d_block_sums[0], d_block_sums[1], blockIdx.x);

	//unsigned int d_in_val_0 = 0;
	//unsigned int d_in_val_1 = 0;

	// Simple implementation's performance is not significantly (if at all)
	//  better than previous verbose implementation
	unsigned int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    //printf("(%d %d %d) --> %d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x, cpy_idx, numElems, d_block_sum_val);
	if (cpy_idx < numElems)
	{
		d_out[cpy_idx] += d_block_sum_val;
		if (cpy_idx + blockDim.x < numElems) {
			d_out[cpy_idx + blockDim.x] +=  d_block_sum_val;
	    }
	}

	//if (2 * glbl_t_idx < numElems)
	//{
	//	d_out[2 * glbl_t_idx] = d_in[2 * glbl_t_idx] + d_block_sum_val;
	//	if (2 * glbl_t_idx + 1 < numElems)
	//		d_out[2 * glbl_t_idx + 1] = d_in[2 * glbl_t_idx + 1] + d_block_sum_val;
	//}

	//if (2 * glbl_t_idx < numElems)
	//{
	//	d_in_val_0 = d_in[2 * glbl_t_idx];
	//	if (2 * glbl_t_idx + 1 < numElems)
	//		d_in_val_1 = d_in[2 * glbl_t_idx + 1];
	//}
	//else
	//	return;
	//__syncthreads();

	//d_out[2 * glbl_t_idx] = d_in_val_0 + d_block_sum_val;
	//if (2 * glbl_t_idx + 1 < numElems)
	//	d_out[2 * glbl_t_idx + 1] = d_in_val_1 + d_block_sum_val;
}



}